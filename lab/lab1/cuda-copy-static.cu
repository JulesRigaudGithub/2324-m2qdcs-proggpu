#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

#define N 1024

// Define an static array dA[N] of floats on the GPU
// Definir un tableau de float dA[N] de taille statique sur le GPU
__device__ float dA[N];

int main() {
  float A[N], B[N];
  int i;

  // Initialization
  // Initialisation
  for (i = 0; i < N; i++) { A[i] = (float)i; }

  // cudaMemcpy from A[N] to dA[N]
  // cudaMemcpy de A[N] vers dA[N]
  hipMemcpyToSymbol(HIP_SYMBOL(dA), A, N*sizeof(float), 0, hipMemcpyHostToDevice);

  // cudaMemcpy from dA[N] to B[N]
  // cudaMemcpy de dA[N] vers B[N]
  hipMemcpyFromSymbol(B, HIP_SYMBOL(dA), N*sizeof(float), 0, hipMemcpyDeviceToHost);

  // Wait for GPU kernels to terminate
  // Attendre que les kernels GPUs terminent
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verify the results
  // Verifier le resultat
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "The copy is incorrect!\n"; }
  else { cout << "The copy is correct!\n"; }

  return 0;
}
