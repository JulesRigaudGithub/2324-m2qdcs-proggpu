#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"

using namespace std;

int main(int argc, char **argv) {
  float *A, *B, *dA;
  int N, i;

  if (argc < 2) {
    printf("Usage: %s N\n", argv[0]);
    return 0;
  }
  N = atoi(argv[1]);

  // Initialization
  // Initialisation
  A = (float *) malloc(sizeof(float) * N);
  B = (float *) malloc(sizeof(float) * N);
  for (i = 0; i < N; i++) { A[i] = (float)i; }
  
  // Allocate the dynamic float array dA[N] on the GPU using cudaMalloc
  // Allouer le tableau dA dynamique de taille N sur le GPU avec cudaMalloc 
  hipError_t cuStat;
  cuStat = hipMalloc((void **)&dA, N*sizeof(float));
  if (cuStat != hipSuccess) {
    printf("L'allocation de la memoire a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cuStat));
  }

  // cudaMemcpy from A[N] to dA[N]
  // cudaMemcpy de A[N] vers dA[N]
  hipMemcpy(dA, A, N*sizeof(float), hipMemcpyHostToDevice);

  // cudaMemcpy from dA[N] to B[N]
  // cudaMemcpy de dA[N] vers B[N]
  hipMemcpy(B, dA, N*sizeof(float), hipMemcpyDeviceToHost);

  // Desaollouer le tableau dA[N] sur le GPU
  cuStat = hipFree(dA);
  if (cuStat != hipSuccess) {
    printf("La libération de la memoire a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cuStat));
  }

  // Attendre que les kernels GPUs terminent
  hipError_t cudaerr = hipDeviceSynchronize();
  if (cudaerr != hipSuccess) {
    printf("L'execution du kernel a echoue avec le code d'erreur \"%s\".\n", hipGetErrorString(cudaerr));
  }

  // Verify the result
  // Verifier le resultat
  for (i = 0; i < N; i++) { if (A[i] != B[i]) { break; } }
  if (i < N) { cout << "La copie est incorrecte!\n" << i; }
  else { cout << "La copie est correcte!\n"; }
  free(A);
  free(B);

  return 0;
}
